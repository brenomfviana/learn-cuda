#include "hip/hip_runtime.h"
/*
 * This code performs a simple matrix multiplication without memory sharing.
 *
 * Code adapted from: Matrix Multiplication with CUDA — A basic introduction to
 * the CUDA programming model, link:
 * https://www.shodor.org/media/content/petascale/materials/UPModules/matrixMultiplication/moduleDocument.pdf
 *
 * To run: ./matrix_multiplication.x A_B_HEIGHT A_WIDTH B_WIDTH MAX_RANDOM_VALUE
 *
 * @author Breno Viana
 * @version 29/09/2017
 */
#include <ctime>
#include <cstdlib>
#include "error_checking.cuh"

// Thread block size
#define BLOCK_SIZE 16

/*!
 * Matrices are stored in row-major order:
 * M(row, col) = *(M.elements + row * M.width + col).
 */
typedef struct {
    int width;
    int height;
    float* elements;
} Matrix;

/*!
 * Apply matrix multiplication on GPU.
 *
 * @param A Matrix A
 * @param B Matrix B
 * @param C Resulting matrix
 */
__global__ void mmd__(Matrix A, Matrix B, Matrix C) {
    // Element of the matrix C
    float e = 0.0;
    // Get matrix row
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    // Get matrix column
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    // Check if row and column are not valid
    if (row > C.height || col > C.width) {
        return;
    }
    // Multiplication
    for (int i = 0; i < A.width; ++i) {
        e += (A.elements[row * A.width + i]) * (B.elements[i * B.width + col]);
        C.elements[row * C.width + col] = e;
    }
}

/*!
 * Prepares matrix multiplication on GPU.
 *
 * @param A Matrix A
 * @param B Matrix B
 * @param C Resulting matrix
 */
void matrix_multiplication(const Matrix A, const Matrix B, Matrix C) {
    // Device matrices
    Matrix d_a, d_b, d_c;
    // Load A to device memory
    d_a.height  = A.height;
    d_a.width   = A.width;
    size_t size = A.width * A.height * sizeof(float);
    CudaSafeCall(hipMalloc(&d_a.elements, size));
    CudaSafeCall(hipMemcpy(d_a.elements, A.elements, size,
                 hipMemcpyHostToDevice));
    // Load B to device memory
    d_b.height = B.height;
    d_b.width  = B.width;
    size       = B.width * B.height * sizeof(float);
    CudaSafeCall(hipMalloc(&d_b.elements, size));
    CudaSafeCall(hipMemcpy(d_b.elements, B.elements, size,
                            hipMemcpyHostToDevice));
    // Allocate C in device memory
    d_c.height = C.height;
    d_c.width  = C.width;
    size       = C.width * C.height * sizeof(float);
    CudaSafeCall(hipMalloc(&d_c.elements, size));
    // Blocks per grid
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    // Threads per block
    dim3 dimGrid((B.width  + dimBlock.x - 1) / dimBlock.x,
                 (A.height + dimBlock.y - 1) / dimBlock.y);
    mmd__<<<dimGrid, dimBlock>>>(d_a, d_b, d_c);
    CudaCheckError();
    CudaSafeCall(hipDeviceSynchronize());
    // Read C from device memory
    CudaSafeCall(hipMemcpy(C.elements, d_c.elements, size,
                 hipMemcpyDeviceToHost));
    // Free device memory
    CudaSafeCall(hipFree(d_a.elements));
    CudaSafeCall(hipFree(d_b.elements));
    CudaSafeCall(hipFree(d_c.elements));
}

int main(int argc, char* argv[]) {
    // Create matrices
    Matrix A, B, C;
    // Initialize matrix A
    A.height   = atoi(argv[1]); // Height of A
    A.width    = atoi(argv[2]); // Width of A
    A.elements = (float*) malloc(A.width * A.height * sizeof(float));
    // Initialize matrix B
    B.height   = A.height;      // Height of B
    B.width    = atoi(argv[3]); // Width of B
    B.elements = (float*) malloc(B.width * B.height * sizeof(float));
    // Initialize matrix C
    C.height   = A.height;
    C.width    =  B.width;
    C.elements = (float*) malloc(C.width * C.height * sizeof(float));
    // Max random value
    int r = atoi(argv[4]);
    srand(time(NULL));
    // Initialize values of matrix A
    for(int i = 0; i < A.height; i++) {
        for(int j = 0; j < A.width; j++) {
            A.elements[i * A.height + j] = rand() % r;
        }
    }
    // Initialize values of matrix B
    for(int i = 0; i < B.height; i++) {
        for(int j = 0; j < B.width; j++) {
            B.elements[i * B.height + j] = rand() % r;
        }
    }

    // Performs matrix calculation (ON GPU)
    matrix_multiplication(A, B, C);

    // Print matrix A
    std::cout << "Matrix A" << std::endl;
    for(int i = 0; i < A.height; i++) {
        for(int j = 0; j < A.width; j++) {
            std::cout << A.elements[i * A.height + j] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
    // Print matrix B
    std::cout << "Matrix B\n" << std::endl;
    for(int i = 0; i < B.height; i++) {
        for(int j = 0; j < B.width; j++) {
            std::cout << B.elements[i * B.height + j] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
    // Print result (matrix C)
    std::cout << "Result of matrix multiplication" << std::endl
              << "Matrix C" << std::endl;
    for(int i = 0; i < C.height; i++) {
        for(int j = 0; j < C.width; j++) {
            std::cout << C.elements[i * C.height + j] << " ";
        }
        std::cout << std::endl;
    }
    // Program successfully completed
    return EXIT_SUCCESS;
}
